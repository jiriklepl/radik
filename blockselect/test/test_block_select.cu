#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include <chrono>
#include <algorithm>
#include <iostream>
#include <thread>

#include "../src/BlockSelectKernel.cuh"
#include "zipf.hpp"

#include <hiprand.h>

// #define CORRECTNESS_CHECK

static constexpr bool LARGEST = 0;

template <typename T>
void generate_uniform_val(T *valIn, int len, T lowerBound, T upperBound) {
    std::uniform_real_distribution<T> dis(lowerBound, upperBound);
    std::default_random_engine generator;
    generator.seed(1);
    std::generate(valIn, valIn + len, [&]() { return dis(generator);});
}

template <typename T>
void generate_zipf_val(T *valIn, int len, double skew, double scale = 1.0) {
    std::default_random_engine generator;
    generator.seed(1);
    ZipfRejectionSampler<std::default_random_engine> zipf(generator, len, skew);
    std::generate(valIn, valIn + len, [&]() { return static_cast<T>(zipf.getSample() * scale);});
}

template <typename IdxType, typename ValType>
void profBlockSelect(const int BATCHSIZE,
                     const int N,
                     const int K,
                     const int DISTRIBUTION_TYPE = 0) {
    // prepare data CPU
    std::vector<int> TASKLEN(BATCHSIZE, N);
    const int64_t TOTALLEN = std::accumulate(TASKLEN.begin(), TASKLEN.end(), 0);
    const int64_t MAXLEN = *std::max_element(TASKLEN.begin(), TASKLEN.end());
    std::vector<int64_t> TASKOFFSET(BATCHSIZE + 1, 0);
    for (int i = 0; i < BATCHSIZE; ++i) {
        TASKOFFSET[i + 1] = TASKLEN[i] + TASKOFFSET[i];
    }

    // prepare data GPU
    ValType *valIn_dev = 0;
    hipMalloc(&valIn_dev, sizeof(ValType) * BATCHSIZE * N);

    std::vector<ValType> valIn(TOTALLEN);
    if (DISTRIBUTION_TYPE == 0) {
        // using U[0, 1]
        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
        hiprandGenerateUniform(gen, valIn_dev, TOTALLEN);
#ifdef CORRECTNESS_CHECK
        hipMemcpy(valIn.data(), valIn_dev, sizeof(ValType) * TOTALLEN, hipMemcpyDeviceToHost);
#endif
        hiprandDestroyGenerator(gen);
    } else if (DISTRIBUTION_TYPE == 1) {
        // using U[0.6, 0.7]
        generate_uniform_val<ValType>(valIn.data(), TOTALLEN, 0.6, 0.7);
        hipMemcpy(valIn_dev, valIn.data(), sizeof(ValType) * TOTALLEN, hipMemcpyDefault);
    } else if (DISTRIBUTION_TYPE == 2) {
        // using U[128.6, 128.7]
        generate_uniform_val<ValType>(valIn.data(), TOTALLEN, 128.6, 128.7);
        hipMemcpy(valIn_dev, valIn.data(), sizeof(ValType) * TOTALLEN, hipMemcpyDefault);
    } else if (DISTRIBUTION_TYPE == 3) {
        std::vector<std::thread> thds;
        // using Zipf(N, 1.1)
        for (int i = 0; i < BATCHSIZE; i++) {
            thds.emplace_back([&](int idx) {
                generate_zipf_val<ValType>(valIn.data() + TASKOFFSET[idx], TASKLEN[idx], 1.1, 1.0 / TASKLEN[idx]);
            }, i);
        }
        for (auto& t: thds) {
            t.join();
        }
        hipMemcpy(valIn_dev, valIn.data(), sizeof(ValType) * TOTALLEN, hipMemcpyDefault);
    } else if (DISTRIBUTION_TYPE == 4) {
        // all zero
        hipMemset(valIn_dev, 0, sizeof(ValType) * TOTALLEN);
    } else {
        throw std::runtime_error("Bad distributiion");
    }

    // IdxType *idxIn_dev = 0;
    ValType *valOut_dev = 0;
    hipMalloc(&valOut_dev, sizeof(ValType) * BATCHSIZE * K);
    IdxType *idxOut_dev = 0;
    hipMalloc(&idxOut_dev, sizeof(IdxType) * BATCHSIZE * K);

    float time = 0.;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    faiss::gpu::runBlockSelect(valIn_dev,
                               valOut_dev,
                               idxOut_dev,
                               LARGEST,
                               K,
                               BATCHSIZE,
                               N,
                               0);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);


    printf("elapsed: %f ms\n", time);

#ifdef CORRECTNESS_CHECK
    std::vector<ValType> res(BATCHSIZE * K);
    hipMemcpy(res.data(), valOut_dev, sizeof(ValType) * BATCHSIZE * K, hipMemcpyDeviceToHost);
    int offset = LARGEST ? N - K : 0;
    for (int i = 0; i < BATCHSIZE; ++i) {
        std::sort(res.begin() + i * K, res.begin() + (i + 1) * K);
        std::sort(dataIn.begin() + i * N, dataIn.begin() + (i + 1) * N);
        for (int j = 0; j < K; ++j) {
            if (dataIn[i * N + j + offset] != res[i * K + j]) {
                std::cout<<"error at ["<<i<<", "<<j<<"], CPU:"<<dataIn[i * N + j + offset]<<", GPU:"<<res[i * K + j]<<std::endl;
            }
        }
    }
#endif
}

int main(int argc, char *argv[]) {
    int BATCHSIZE;
    int K;
    int N;
    int DISTRIBUTION = 0;

    if (argc >= 4) {
        BATCHSIZE = atoi(argv[1]);
        N = (1<<atoi(argv[2]));
        K = atoi(argv[3]);
        if (argc >= 5) {
            DISTRIBUTION = atoi(argv[4]);
        }
    } else {
        printf("Please enter BATCHSIZE:\n");
        std::cin>>BATCHSIZE;
        printf("Please enter power of N(max val 29):\n");
        std::cin>>N;
        N = (1<<N);
        printf("Please enter K:\n");
        std::cin>>K;
    }

    profBlockSelect<int32_t, float>(BATCHSIZE, N, K, DISTRIBUTION);

    return 0;
}
